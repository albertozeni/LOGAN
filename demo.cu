
//=========================================================
// Title:	LOGAN's Demo
// Authors:	A.Zeni, G. Guidi
//=========================================================

#include <omp.h>
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>
#include <cstdlib>
#include <cstdio>
#include <algorithm>
#include <cmath>
#include <numeric>
#include <vector>
#include <sys/types.h>
#include <sys/stat.h>
#include <math.h>
#include <limits.h>
#include <bitset>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <ctype.h>
#include <sstream>
#include <cassert>
#include <set>
#include <memory>
#include <typeinfo>
#include <vector>

#ifdef __NVCC__
	#include "src/gpu/coalesced/logan.cuh"
#endif

#define BATCH_SIZE 30000

/* nucleotide complement */
char basecomplement (char n)
{
	switch(n)
	{
	case 'A':
		return 'T';
	case 'T':
		return 'A';
	case 'G':
		return 'C';
	case 'C':
		return 'G';
	}
	assert(false);
	return ' ';
}

std::vector<std::string> split(const std::string &s, char delim)
{
	std::vector<std::string> result;
	std::stringstream ss(s);
	std::string item;

	while (std::getline(ss, item, delim))
		result.push_back(item);

	return result;
}

/* LOGAN's function call */
void LOGAN(std::vector<std::vector<std::string>> &alignments, int ksize,
				int xdrop, int AlignmentsToBePerformed, int ngpus, int maxt)
{
	std::vector<int> 	posV(AlignmentsToBePerformed);
	std::vector<int> 	posH(AlignmentsToBePerformed);
	std::vector<SeedL> 	seeds(AlignmentsToBePerformed);
	std::vector<std::string> seqsV(AlignmentsToBePerformed);
	std::vector<std::string> seqsH(AlignmentsToBePerformed);
	std::vector<ScoringSchemeL> penalties(AlignmentsToBePerformed);
	ScoringSchemeL sscheme(1, -1, -1, -1);

	/* Pre-processing */
	for(int i = 0; i < AlignmentsToBePerformed; i++)
	{
        posV[i]	 =	std::stoi(alignments[i][1]);
        posH[i]	 =	std::stoi(alignments[i][3]);
        seqsV[i] =	alignments[i][0];
        seqsH[i] =	alignments[i][2];
        std::string strand = alignments[i][4];

		/* Reverse complement */
        if(strand == "c")
		{
            std::transform(
                std::begin(seqsH[i]),
                std::end(seqsH[i]),
                std::begin(seqsH[i]),
                basecomplement);
            posH[i] = seqsH[i].length()-posH[i]-ksize;
        }

		/* match, mismatch, gap opening, gap extension */
		penalties[i] = sscheme;
		/* starting position on seqsH, starting position on seqsV, k-mer/seed size */
		SeedL sseed(posH[i], posV[i], ksize);
		seeds[i] = sseed;
    }

	int numAlignmentsLocal = BATCH_SIZE * ngpus;
	cout <<"///////////////////////////////////////////////" << ngpus << endl;

	//	Divide the alignment in batches of 30K alignments
	for(int i = 0; i < AlignmentsToBePerformed; i += BATCH_SIZE * ngpus)
	{
		if(AlignmentsToBePerformed < (i + BATCH_SIZE * ngpus))
			numAlignmentsLocal = AlignmentsToBePerformed % (BATCH_SIZE * ngpus);

		int* res = (int*)malloc(numAlignmentsLocal * sizeof(int));

		std::vector<string>::const_iterator first_t = seqsH.begin() + i;
		std::vector<string>::const_iterator last_t  = seqsH.begin() + i + numAlignmentsLocal;
		std::vector<string> target_b(first_t, last_t);

		std::vector<string>::const_iterator first_q = seqsV.begin() + i;
		std::vector<string>::const_iterator last_q  = seqsV.begin() + i + numAlignmentsLocal;
		std::vector<string> query_b(first_q, last_q);

		std::vector<SeedL>::const_iterator first_s = seeds.begin() + i;
		std::vector<SeedL>::const_iterator last_s  = seeds.begin() + i + numAlignmentsLocal;
		std::vector<SeedL> seeds_b(first_s, last_s);

		extendSeedL(seeds_b, EXTEND_BOTHL, target_b, query_b, penalties, xdrop, ksize, res, numAlignmentsLocal, ngpus, maxt);
		free(res);
	}
}

int main(int argc, char **argv)
{
	if(argc!=5){
		std::cout<<"Syntax: "<<argv[0]<<" <input> <k-mer-length> <X-drop> <#GPUS>"<<std::endl;
		return -1;
	}

	std::ifstream input(argv[1]);

	int ksize = atoi(argv[2]);
	int xdrop = atoi(argv[3]);
	int ngpus = atoi(argv[4]);

	int maxt = 1;
	#pragma omp parallel
	{
		maxt = omp_get_num_threads();
	}

	/* Init the GPU environment */
	hipFree(0);

	// @AlignmentsToBePerformed = alignments to be performed
	const auto AlignmentsToBePerformed = std::count(std::istreambuf_iterator<char>(input), std::istreambuf_iterator<char>(), '\n');
    input.seekg(0, std::ios_base::beg);

    /* Read input file */
	std::vector<std::string> entries;
    if(input)
        for (int64_t i = 0; i < AlignmentsToBePerformed; ++i)
        {
            std::string line;
            std::getline(input, line);
            entries.push_back(line);
        }
    input.close();

	std::vector<std::vector<std::vector<std::string>>> local(maxt);
	std::vector<std::vector<std::string>> alignments(AlignmentsToBePerformed);

	/* Pre-processing */
	#pragma omp parallel for
    for(int64_t i = 0; i < AlignmentsToBePerformed; i++)
    {
		int tid = omp_get_thread_num();
        std::vector<std::string> tmp = split(entries[i], '\t');
        local[tid].push_back(tmp);
    }

	unsigned int alignmentssofar = 0;
	for(int tid = 0; tid < maxt; ++tid)
	{
		copy(local[tid].begin(), local[tid].end(), alignments.begin() + alignmentssofar);
		alignmentssofar += local[tid].size();
	}

	/* Compute pairwise alignments */
   	LOGAN(alignments, ksize, xdrop, AlignmentsToBePerformed, ngpus, maxt);

   	return 0;
}

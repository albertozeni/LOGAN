
//=========================================================
// Title:	LOGAN's Demo
// Author:	G. Guidi
// Created:	12 March 2019
// Last modified: 18 October 2019
//=========================================================

#include <omp.h>
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>
#include <cstdlib>
#include <cstdio>
#include <algorithm>
#include <cmath>
#include <numeric>
#include <vector>
#include <sys/types.h> 
#include <sys/stat.h> 
#include <math.h>
#include <limits.h>
#include <bitset>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <ctype.h> 
#include <sstream>
#include <cassert>
#include <set>
#include <memory>
#include <typeinfo>
#include <vector>

#ifdef __NVCC__
	#include "src/gpu/coalesced/logan.cuh"
#endif

/* nucleotide complement */
char basecomplement (char n)
{	
	switch(n)
	{   
	case 'A':
		return 'T';
	case 'T':
		return 'A';
	case 'G':
		return 'C';
	case 'C':
		return 'G';
	}	
	assert(false);
	return ' ';
}

std::vector<std::string> split(const std::string &s, char delim)
{
	std::vector<std::string> result;
	std::stringstream ss(s);
	std::string item;

	while (std::getline(ss, item, delim))
		result.push_back(item);

	return result;
}

/* LOGAN's function call */
void LOGAN(std::vector<std::vector<std::string>> &alignments, int ksize, 
				int xdrop, int num, int ngpus, int maxt)
{
	std::vector<int> 	posV(num);
	std::vector<int> 	posH(num);
	std::vector<SeedL> 	seeds(num);
	std::vector<std::string> seqsV(num);
	std::vector<std::string> seqsH(num);
	std::vector<ScoringSchemeL> penalties(num);

	/* Pre-processing */
	for(int i = 0; i < num; i++)
	{
        posV[i]	 =	std::stoi(alignments[i][1]);
        posH[i]	 =	std::stoi(alignments[i][3]);
        seqsV[i] =	alignments[i][0];
        seqsH[i] =	alignments[i][2];
        std::string strand = alignments[i][4];

		/* Reverse complement */
        if(strand == "c")
		{
            std::transform(
                std::begin(seqsH[i]),
                std::end(seqsH[i]),
                std::begin(seqsH[i]),
                basecomplement);
            posH[i] = seqsH[i].length()-posH[i]-ksize;
        }

		/* match, mismatch, gap opening, gap extension */ 
		ScoringSchemeL sscheme(1, -1, -1, -1);
		penalties[i] = sscheme;
		SeedL sseed(posH[i], posV[i], ksize);
		seeds[i] = sseed;
    }

    int* results = (int*)malloc(sizeof(int)*num);
	/* TODO: maxt not used */
	/* Batch alignment on GPU */
    extendSeedL(seeds, EXTEND_BOTHL, seqsH, seqsV, penalties, xdrop, ksize, results, num, ngpus, maxt);
}

int main(int argc, char **argv)
{
	std::ifstream input(argv[1]);

	int ksize = atoi(argv[2]);	
	int xdrop = atoi(argv[3]);	
	int ngpus = atoi(argv[5]);

	int maxt = 1;
	#pragma omp parallel
	{
		maxt = omp_get_num_threads();
	}

	/* Init the GPU environment */
	hipFree(0);

	uint64_t num = std::count(std::istreambuf_iterator<char>(input), std::istreambuf_iterator<char>(), '\n');
    input.seekg(0, std::ios_base::beg);

    /* Read input file */
	std::vector<std::string> entries;
    if(input)
        for (int i = 0; i < num; ++i)
        {
            std::string line;
            std::getline(input, line);
            entries.push_back(line);
        }
    input.close();

	std::vector<std::vector<std::vector<std::string>>> local(maxt);
	std::vector<std::vector<std::string>> alignments(num);

	/* Pre-processing */
	#pragma omp parallel for
    for(uint64_t i = 0; i < num; i++)
    {
		int tid = omp_get_thread_num();
        std::vector<std::string> tmp = split(entries[i], '\t');
        local[tid].push_back(tmp);
    }

	unsigned int alignmentssofar = 0;
	for(int tid = 0; tid < maxt; ++tid)
	{
		copy(local[tid].begin(), local[tid].end(), alignments.begin() + alignmentssofar);
		alignmentssofar += local[tid].size();
	}

	/* Compute pairwise alignments */
   	LOGAN(alignments, ksize, xdrop, num, ngpus, maxt);	
		
   	return 0;
}



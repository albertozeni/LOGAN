
//=========================================================
// Title:	LOGAN's Demo
// Authors:	A.Zeni, G. Guidi
//=========================================================

#include "logan.cuh"

using namespace std;

#define BATCH_SIZE 30000
#define GPU_THREADS 128

/* nucleotide complement */
char basecomplement (char n)
{	
	switch(n)
	{   
	case 'A':
		return 'T';
	case 'T':
		return 'A';
	case 'G':
		return 'C';
	case 'C':
		return 'G';
	}	
	assert(false);
	return ' ';
}

std::vector<std::string> split(const std::string &s, char delim)
{
	std::vector<std::string> result;
	std::stringstream ss(s);
	std::string item;

	while (std::getline(ss, item, delim))
		result.push_back(item);

	return result;
}

/* LOGAN's function call */
void LOGAN(std::vector<std::vector<std::string>> &alignments, int ksize, 
				int xdrop, int AlignmentsToBePerformed, int ngpus, int maxt)
{
	std::vector<int> 	posV(AlignmentsToBePerformed);
	std::vector<int> 	posH(AlignmentsToBePerformed);
	std::vector<SeedL> 	seeds(AlignmentsToBePerformed);
	std::vector<std::string> seqsV(AlignmentsToBePerformed);
	std::vector<std::string> seqsH(AlignmentsToBePerformed);
	std::vector<ScoringSchemeL> penalties(AlignmentsToBePerformed);
	ScoringSchemeL sscheme(1, -1, -1, -1);

	/* Pre-processing */
	for(int i = 0; i < AlignmentsToBePerformed; i++)
	{
        posV[i]	 =	std::stoi(alignments[i][1]);
        posH[i]	 =	std::stoi(alignments[i][3]);
        seqsV[i] =	alignments[i][0];
        seqsH[i] =	alignments[i][2];
        std::string strand = alignments[i][4];

		/* Reverse complement */
        if(strand == "c")
		{
            std::transform(
                std::begin(seqsH[i]),
                std::end(seqsH[i]),
                std::begin(seqsH[i]),
                basecomplement);
            posH[i] = seqsH[i].length()-posH[i]-ksize;
        }

		/* match, mismatch, gap opening, gap extension */ 
		penalties[i] = sscheme;
		/* starting position on seqsH, starting position on seqsV, k-mer/seed size */
		SeedL sseed(posH[i], posV[i], ksize);
		seeds[i] = sseed;
    }

	int numAlignmentsLocal = BATCH_SIZE * ngpus; 
	
	//	Divide the alignment in batches of 30K alignments
	for(int i = 0; i < AlignmentsToBePerformed; i += BATCH_SIZE * ngpus)
	{
		if(AlignmentsToBePerformed < (i + BATCH_SIZE * ngpus))
			numAlignmentsLocal = AlignmentsToBePerformed % (BATCH_SIZE * ngpus);

		int* res = (int*)malloc(numAlignmentsLocal * sizeof(int));	

		std::vector<string>::const_iterator first_t = seqsH.begin() + i;
		std::vector<string>::const_iterator last_t  = seqsH.begin() + i + numAlignmentsLocal;
		std::vector<string> target_b(first_t, last_t);

		std::vector<string>::const_iterator first_q = seqsV.begin() + i;
		std::vector<string>::const_iterator last_q  = seqsV.begin() + i + numAlignmentsLocal;
		std::vector<string> query_b(first_q, last_q);

		std::vector<SeedL>::const_iterator first_s = seeds.begin() + i;
		std::vector<SeedL>::const_iterator last_s  = seeds.begin() + i + numAlignmentsLocal;
		std::vector<SeedL> seeds_b(first_s, last_s);

		extendSeedL(seeds_b, EXTEND_BOTHL, target_b, query_b, penalties, xdrop, ksize, res, numAlignmentsLocal, ngpus, maxt);
		free(res);
	}
}

int main(int argc, char **argv)
{
	std::ifstream input(argv[1]);

	int ksize = atoi(argv[2]);	
	int xdrop = atoi(argv[3]);	
	int ngpus = atoi(argv[4]);

	/* Init the GPU environment */
	hipFree(0);

	// @AlignmentsToBePerformed = alignments to be performed
	uint64_t AlignmentsToBePerformed = std::count(std::istreambuf_iterator<char>(input), std::istreambuf_iterator<char>(), '\n');
    input.seekg(0, std::ios_base::beg);

    /* Read input file */
	std::vector<std::string> entries;
    if(input)
        for (int i = 0; i < AlignmentsToBePerformed; ++i)
        {
            std::string line;
            std::getline(input, line);
            entries.push_back(line);
        }
    input.close();

	std::vector<std::vector<std::vector<std::string>>> local(GPU_THREADS);
	std::vector<std::vector<std::string>> alignments(AlignmentsToBePerformed);

	/* Pre-processing */
	#pragma omp parallel for
    for(uint64_t i = 0; i < AlignmentsToBePerformed; i++)
    {
		int tid = omp_get_thread_num();
        std::vector<std::string> tmp = split(entries[i], '\t');
        local[tid].push_back(tmp);
    }

	unsigned int alignmentssofar = 0;
	for(int tid = 0; tid < GPU_THREADS; ++tid)
	{
		copy(local[tid].begin(), local[tid].end(), alignments.begin() + alignmentssofar);
		alignmentssofar += local[tid].size();
	}

	/* Compute pairwise alignments */
	auto start = NOW;
   	LOGAN(alignments, ksize, xdrop, AlignmentsToBePerformed, ngpus, GPU_THREADS);	
	auto end = NOW;	
	std::chrono::duration<double> tot_time = end-start;
	double duration_tot = tot_time.count();


	std::cout<< "Total Execution time:\t"<< duration_tot <<std::endl;
   	return 0;
}



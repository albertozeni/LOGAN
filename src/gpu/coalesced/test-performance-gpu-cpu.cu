
//========================================================================================================
// Title:  C++ program to assest quality and performance of LOGAN wrt to original SeqAn implementation
// Author: G. Guidi
// Date:   12 March 2019
//========================================================================================================

//#include <omp.h>
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>
#include <cstdlib>
#include <cstdio>
#include <algorithm>
#include <cmath>
#include <numeric>
#include <vector>
#include <sys/types.h> 
#include <sys/stat.h> 
#include <math.h>
#include <limits.h>
#include <bitset>
#include <map>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <ctype.h> 
#include <sstream>
#include <set>
#include <memory>
#include <typeinfo>
#include <pthread.h>
#include <vector>
#include <seqan/sequence.h>
#include <seqan/align.h>
#include <seqan/seeds.h>
#include <seqan/score.h>
#include <seqan/modifier.h>
#include"logan.cuh"

using namespace std;
//using namespace seqan;

#define NOW std::chrono::high_resolution_clock::now()

//=======================================================================
// 
// Common functions
// 
//=======================================================================

//typedef seqan::Seed<seqan::Simple> TSeed;
typedef std::tuple< int, int, int, int, int, double > myinfo;	// score, start seedV, end seedV, start seedH, end seedH, runtime

char dummycomplement (char n)
{	
	switch(n)
	{   
	case 'A':
		return 'T';
	case 'T':
		return 'A';
	case 'G':
		return 'C';
	case 'C':
		return 'G';
	}	
	assert(false);
	return ' ';
}

vector<std::string> split (const std::string &s, char delim)
{
	std::vector<std::string> result;
	std::stringstream ss (s);
	std::string item;

	while (std::getline (ss, item, delim))
	{
		result.push_back (item);
	}

	return result;
}

//=======================================================================
// 
// SeqAn and LOGAN function calls
// 
//=======================================================================

typedef seqan::Seed<seqan::Simple> TSeed;
void loganXdrop(std::vector< std::vector<std::string> > &v, int mat, int mis, int gap, int kmerLen, int xdrop, int numpair, int gpus, int n_threads)
{
	
	
	//Result result(kmerLen);
	int n_align = v.size();
	//int result;
	//myinfo loganresult;
	vector<ScoringSchemeL> penalties(n_align);
	vector<int> posV(n_align);
	vector<int> posH(n_align);
	vector<string> seqV(n_align);
	vector<string> seqH(n_align);
	vector<SeedL> seeds(n_align);
	for(int i = 0; i < v.size(); i++){
                ScoringSchemeL tmp_sscheme(mat, mis, -1, gap);
                penalties[i]=tmp_sscheme;
                posV[i]=stoi(v[i][1]);
                posH[i]=stoi(v[i][3]);
                seqV[i]=v[i][0];
                seqH[i]=v[i][2];
                std::string strand = v[i][4];

                if(strand == "c"){
                        std::transform(
                                        std::begin(seqH[i]),
                                        std::end(seqH[i]),
                                        std::begin(seqH[i]),
                                        dummycomplement);
                        posH[i] = seqH[i].length()-posH[i]-kmerLen;
                }
		SeedL tmp_seed(posH[i], posV[i], kmerLen);
		seeds[i] = tmp_seed;
        }
	//seqan testbench
	seqan::Score<int, seqan::Simple> scoringScheme_s(mat, mis, -1, gap);
        cout<< "PERFORMING "<< numpair << " ALIGNMENTS"<<endl;
        int *scoreSeqAn;
	scoreSeqAn = (int *)malloc(sizeof(int)*numpair);
        std::cout << "STARTING CPU" << std::endl;
        std::chrono::duration<double>  diff_s;
        vector<seqan::Dna5String> seqV_s_arr(numpair);
	vector<seqan::Dna5String> seqH_s_arr(numpair);
	vector<TSeed> seed(numpair);
        for(int i = 0; i < numpair; i++){
                seqan::Dna5String seqV_s(seqV[i]);
                seqan::Dna5String seqH_s(seqH[i]);
                seqV_s_arr[i]=seqV_s;
                seqH_s_arr[i]=seqH_s;
                TSeed tmp(posH[i], posV[i], kmerLen);
                seed[i]=tmp;
        }
        auto start_s = std::chrono::high_resolution_clock::now();
        for(int i = 0; i < numpair; i++){
                scoreSeqAn[i] = seqan::extendSeed(seed[i], seqH_s_arr[i], seqV_s_arr[i], seqan::EXTEND_BOTH, scoringScheme_s, xdrop, seqan::GappedXDrop(), kmerLen);
        }
        auto end_s = std::chrono::high_resolution_clock::now();
        diff_s = end_s-start_s;
        cout << "SEQAN TIME:\t" <<  diff_s.count() <<endl;

        int *scoreLogan;
	scoreLogan = (int *)malloc(sizeof(int)*numpair);
        std::chrono::duration<double>  diff_l;
        std::cout << "STARTING GPU" << std::endl;
        auto start_l = NOW;
        extendSeedL(seeds, EXTEND_BOTHL, seqH, seqV, penalties, xdrop, kmerLen, scoreLogan, numpair, gpus, n_threads);
        auto end_l = NOW;
        diff_l = end_l-start_l;

        cout << "LOGAN TIME:\t" <<  diff_l.count() <<endl;
        cout << "CHECKING RESULTS"<< endl;
        bool test = true;
        for(int i = 0; i<numpair; i++){
                if(scoreLogan[i]!=scoreSeqAn[i]){
                        test = false;
                        cout << "ERROR ALIGNMENT: "<< i << endl;
                        cout << "SEQAN ALIGNMENT: "<< scoreSeqAn[i] << " LOGAN ALIGNMENT: " << scoreLogan[i] << endl;
                }
		else if(seqan::endPositionH(seed[i])!=seeds[i].endPositionH||seqan::endPositionV(seed[i])!=seeds[i].endPositionV||seqan::beginPositionV(seed[i])!=seeds[i].beginPositionV||seqan::beginPositionH(seed[i])!=seeds[i].beginPositionH){
			test = false;
			cout << "ERROR SEED: "<< i << endl;
                        cout << "ENDH SEQAN: "<< seqan::endPositionH(seed[i]) << " LOGAN: " << seeds[i].endPositionH << endl;
			cout << "ENDV SEQAN: "<< seqan::endPositionV(seed[i]) << " LOGAN: " << seeds[i].endPositionV << endl;
			cout << "BEGINH SEQAN: "<< seqan::beginPositionH(seed[i]) << " LOGAN: " << seeds[i].beginPositionH << endl;
			cout << "BEGINV SEQAN: "<< seqan::beginPositionV(seed[i]) << " LOGAN: " << seeds[i].beginPositionV << endl;
		}
        }
        if(test){
                cout << "ALL OK\n" << "SPEEDUP " << diff_s.count()/diff_l.count()<<"X"<< endl;
	}
	else{
		cout << "ERROR BUT..\n" << "SPEEDUP " << diff_s.count()/diff_l.count()<<"X"<< endl;
	}
}

//=======================================================================
//
// Function call main
//
//=======================================================================

int main(int argc, char **argv)
{
	// add optlist library later		
	ifstream input(argv[1]);		// file name with sequences and seed positions
	int kmerLen = atoi(argv[2]);	// kmerLen
	int xdrop = atoi(argv[3]);		// xdrop
	int n_threads = atoi(argv[4]);
	int gpus = atoi(argv[5]);
	int mat = 1, mis = -1, gap = -1;	// GGGG: make these input parameters
	const char* filename =  (char*) malloc(20 * sizeof(char));
	//filename = temp.c_str();
	std::cout << "STARTING BENCHMARK" << std::endl;
	
	//setting up the gpu environment
	hipFree(0);

	uint64_t numpair = std::count(std::istreambuf_iterator<char>(input), std::istreambuf_iterator<char>(), '\n');
        input.seekg(0, std::ios_base::beg);

        vector<std::string> entries;

        /* read input file */
        if(input)
                for (int i = 0; i < numpair; ++i)
                {
                        std::string line;
                        std::getline(input, line);
                        entries.push_back(line);
                }
        input.close();
        // compute pairwise alignments
	vector< vector<string> > v(numpair);
        for(uint64_t i = 0; i < numpair; i++)
        {

                //int ithread = i;//omp_get_thread_num();
                vector<string> temp = split(entries[i], '\t');
                // format: seqV, posV, seqH, posH, strand -- GGGG: generate this input with BELLA
                v[i]=temp;
        }
        loganXdrop(v, mat, mis, gap, kmerLen, xdrop, numpair, gpus, n_threads);	

	return 0;
}

